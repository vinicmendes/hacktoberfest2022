
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_ITER 100
#define MAX 100 // maximum value of the matrix element
#define ERROR 0.000001

// Generate a random float number with the maximum value of max
float rand_float(int max)
{
    return ((float)rand() / (float)(RAND_MAX)) * max;
}

// Allocate 2D matrix
void allocate_init_2Dmatrix(float ***mat, int n, int m)
{
    int i, j;
    *mat = (float **)malloc(n * sizeof(float *));
    for (i = 0; i < n; i++)
    {
        (*mat)[i] = (float *)malloc(m * sizeof(float));
        for (j = 0; j < m; j++)
            (*mat)[i][j] = rand_float(MAX);
    }
}

// solver
void solver(float ***mat, int n, int m)
{
    float diff = 0, temp;
    int done = 0, cnt_iter = 0, i, j;

    while (!done && (cnt_iter < MAX_ITER))
    {
        diff = 0;
        for (i = 1; i < n - 1; i++)
            for (j = 1; j < m - 1; j++)
            {
                temp = (*mat)[i][j];
                (*mat)[i][j] = 0.2 * ((*mat)[i][j] + (*mat)[i][j - 1] + (*mat)[i - 1][j] + (*mat)[i][j + 1] + (*mat)[i + 1][j]);
                diff += abs((*mat)[i][j] - temp);
            }
        if (diff / n / n < ERROR)
            done = 1;
        cnt_iter++;
    }

    if (done)
        printf("Convergiu após %d iterações\n", cnt_iter);
    else
        printf("Não convergiu após %d iterações\n", cnt_iter);
}

int main(int argc, char *argv[])
{
    int n, communication;
    float **a;

    if (argc < 2)
    {
        printf("Passe dois parâmetros: matrix_size communication \n");
        printf("\t matrix_size: Adicione 2 a uma potência de 2 (e.g. : 18, 1026)\n");

        exit(1);
    }

    n = atoi(argv[1]);
    printf("Tamanho da matriz = %d\n", n);

    allocate_init_2Dmatrix(&a, n, n);

    solver(&a, n, n);

    return 0;
}
